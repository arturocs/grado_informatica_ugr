#include "hip/hip_runtime.h"
#include "Graph.h"
#include <fstream>
#include <iostream>
#include <string.h>
#include <sys/time.h>

// CUDA runtime
//#include <hip/hip_runtime.h>
// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>


#define blocksize2d 32
#define blocksize blocksize2d * blocksize2d

using namespace std;

//**************************************************************************

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

//**************************************************************************

__global__ void floyd_kernel(int *M, const int nverts, const int k) {
  int ij = threadIdx.x + blockDim.x * blockIdx.x;
  if (ij < nverts * nverts) {
    int Mij = M[ij];
    int i = ij / nverts;
    int j = ij - i * nverts;
    if (i != j && i != k && j != k) {
      int Mikj = M[i * nverts + k] + M[k * nverts + j];
      Mij = (Mij > Mikj) ? Mikj : Mij;
      M[ij] = Mij;
    }
  }
}

__global__ void reduction(int *M, int *out, int n_elem) {

  extern __shared__ int sdata[];
  int ij = blockIdx.x * blockDim.x + threadIdx.x;
  if (ij < n_elem) {
    int n, m, tid = threadIdx.x;
    sdata[tid] = M[ij];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
      if (tid < s) {
        n = sdata[tid];
        m = sdata[tid + s];
        sdata[tid] = max(n, m);
      }
      __syncthreads();
    }
  }

  if (threadIdx.x == 0)
    out[blockIdx.x] = sdata[0];
}

__global__ void floyd_kernel_2d(int *M, const int nverts, const int k) {
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  int i = threadIdx.y + blockDim.y * blockIdx.y;
  int index = i * nverts + j;
  if (i < nverts && j < nverts) {
    int Mij = M[index];
    if (i != j && i != k && j != k) {
      int Mikj = M[i * nverts + k] + M[k * nverts + j];
      Mij = (Mij > Mikj) ? Mikj : Mij;
      M[index] = Mij;
    }
  }
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    cerr << "Sintaxis: " << argv[0] << " <archivo de grafo>" << endl;
    return (-1);
  }

  // This will pick the best possible CUDA capable device
  // int devID = findCudaDevice(argc, (const char **)argv);
  // Get GPU information

  int devID;
  hipDeviceProp_t props;
  hipError_t err;

  err = hipGetDevice(&devID);

  if (err != hipSuccess) {
    cout << "ERRORRR" << endl;
  }

  hipGetDeviceProperties(&props, devID);

  printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name,
         props.major, props.minor);

  Graph G;

  G.lee(argv[1]); // Read the Graph

  // cout << "EL Grafo de entrada es:"<<endl;

  const int nverts = G.vertices;
  const int niters = nverts;
  const int nverts2 = nverts * nverts;

  int *c_Out_M = new int[nverts2];
  int *c_Out_M_2d = new int[nverts2];
  int size = nverts2 * sizeof(int);
  int *d_In_M = NULL;
  int *d_In_M_2d = NULL;
  int *A = G.Get_Matrix();
  int *v;
  int *v_d;

  err = hipMalloc((void **)&d_In_M, size);
  err = hipMalloc((void **)&d_In_M_2d, size);

  if (err != hipSuccess) {
    cout << "ERROR RESERVA" << endl;
  }

  // GPU phase

  err = hipMemcpy(d_In_M, A, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "ERROR COPIA A GPU" << endl;
  }

  double t1 = cpuSecond();

  // kernel 1d

  for (int k = 0; k < niters; k++) {
    // printf("CUDA kernel launch \n");
    int threadsPerBlock = blocksize;
    int blocksPerGrid = (nverts2 + threadsPerBlock - 1) / threadsPerBlock;

    floyd_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_In_M, nverts, k);
    err = hipGetLastError();

    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel 1d!\n");
      exit(EXIT_FAILURE);
    }
  }

  hipDeviceSynchronize();
  double Tgpu = cpuSecond() - t1;
  hipMemcpy(c_Out_M, d_In_M, size, hipMemcpyDeviceToHost);

  cout << "Tiempo gastado GPU 1D= " << Tgpu << endl << endl;

  // kernel 2d

  err = hipMemcpy(d_In_M_2d, A, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "ERROR COPIA A GPU" << endl;
  }
  t1 = cpuSecond();
  for (int k = 0; k < niters; k++) {
    // printf("CUDA kernel launch \n");
    dim3 threadsPerBlock(blocksize2d, blocksize2d);
    dim3 blocksPerGrid(ceil((float)(nverts) / threadsPerBlock.x),
                       ceil((float)(nverts) / threadsPerBlock.y));

    floyd_kernel_2d<<<blocksPerGrid, threadsPerBlock>>>(d_In_M_2d, nverts, k);

    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel 2d!\n");
      exit(EXIT_FAILURE);
    }
  }

  hipDeviceSynchronize();
  double Tgpu2d = cpuSecond() - t1;

  hipMemcpy(c_Out_M_2d, d_In_M_2d, size, hipMemcpyDeviceToHost);

  cout << "Tiempo gastado GPU 2D= " << Tgpu2d << endl << endl;

  // CPU phase

  t1 = cpuSecond();
  // BUCLE PPAL DEL ALGORITMO
  int inj, in, kn;
  for (int k = 0; k < niters; k++) {
    kn = k * nverts;
    for (int i = 0; i < nverts; i++) {
      in = i * nverts;
      for (int j = 0; j < nverts; j++)
        if (i != j && i != k && j != k) {
          inj = in + j;
          A[inj] = min(A[in + k] + A[kn + j], A[inj]);
        }
    }
  }

  double t2 = cpuSecond() - t1;
  cout << "Tiempo gastado CPU= " << t2 << endl << endl;
  cout << "Ganancia= " << t2 / Tgpu << endl;
  cout << "Ganancia 2d= " << t2 / Tgpu2d << endl;
  for (int i = 0; i < nverts; i++)
    for (int j = 0; j < nverts; j++)
      if (abs(c_Out_M[i * nverts + j] - G.arista(i, j)) > 0)
        cout << "Error (" << i << "," << j << ")   " << c_Out_M[i * nverts + j]
             << "..." << G.arista(i, j) << endl;

  for (int i = 0; i < nverts; i++)
    for (int j = 0; j < nverts; j++)
      if (abs(c_Out_M_2d[i * nverts + j] - G.arista(i, j)) > 0)
        cout << "Error 2d (" << i << "," << j << ")   "
             << c_Out_M_2d[i * nverts + j] << "..." << G.arista(i, j) << endl;

  // Minimum vector  to be computed on GPU

  dim3 threadsPerBlock(blocksize2d, blocksize2d);
  dim3 blocksPerGrid(ceil((int)(nverts) / threadsPerBlock.x),
                     ceil((int)(nverts) / threadsPerBlock.y));

  v = (int *)malloc(blocksPerGrid.x * sizeof(int));
  hipMalloc((void **)&v_d, sizeof(int) * blocksPerGrid.x);

  int smemSize = threadsPerBlock.x * sizeof(int);

  // Kernel launch to compute Minimum Vector

  err = hipMemcpy(d_In_M_2d, c_Out_M_2d, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "ERROR COPIA A GPU" << endl;
  }

  reduction<<<blocksPerGrid, threadsPerBlock, smemSize>>>(d_In_M_2d, v_d,
                                                          nverts * nverts);

  hipMemcpy(v, v_d, blocksPerGrid.x * sizeof(int), hipMemcpyDeviceToHost);

  int max_gpu = 0;
  for (int i = 1; i < blocksPerGrid.x; i++) {
    max_gpu = max(max_gpu, v[i]);
  }

  cout << " Max on GPU =" << max_gpu << endl;

  cout << t2 << "\t"<< Tgpu << "\t" << t2 / Tgpu << "\t"  << Tgpu2d << "\t" << t2 / Tgpu2d << endl;
}
